﻿#include "hip/hip_runtime.h"


#include <stdio.h>

/*
 Miller-Rabin test
 if p = 2 then return (p is a prime)
 if p is even then return (p is not a prime)
 find smallest d such that p = 2^r + d + 1 (d is an odd number)
 randomly choose a such that 2 <= a <= p - 2
 x := a^d mod p
 if x == 1 or x == p - 1 then return (p is probably a prime)
 repeat r - 1 times:
   x := x^2 mod p
   if x == p - 1 then return (p is probably a prime)
 return (p is not a prime)
 There is at most 1/4 numbers a such that the algorithms return false answer.
 The complexity of Miller-Rabin prime test is O(log3p)
 There is also a deterministic polynomial algorithm for the primality test with
 a time complexity O(n12).
*/

hipError_t isPrimeWithCuda(const int *p);

__global__ void millerRabinTestKernel(const int* p, bool* isPrime)
{
    printf("Thread %d: Testing if %d is prime...\n", threadIdx.x, *p);
    int a = threadIdx.x + 2; // Randomly chosen a such that 2 <= a <= p - 2
    int d = *p - 1;
    int r = 0;
    while (d % 2 == 0) {
        d /= 2;
        r++;
    }
    printf("Thread %d: a = %d, d = %d, r = %d\n", threadIdx.x, a, d, r);
    int x = 1;
    for (int i = 0; i < d; i++) {
        x = (x * a) % *p;
    }
    if (x == 1 || x == *p - 1) {
        *isPrime = true;
        printf("Thread %d: %d is probably prime (x == 1 or x == p - 1)\n", threadIdx.x, *p);
        return;
    }
    for (int i = 0; i < r - 1; i++) {
        x = (x * x) % *p;
        if (x == *p - 1) {
            *isPrime = true;
            printf("Thread %d: %d is probably prime (x == p - 1)\n", threadIdx.x, *p);
            return;
        }
    }
    *isPrime = false;
    printf("Thread %d: %d is not prime\n", threadIdx.x, *p);
    return;
}

void isPrimeCpu(int p) {
    if (p < 2) {
        printf("CPU: Input number must be greater than 1!\n");
        return;
    }
    if (p == 2) {
        printf("CPU: Is %d prime? true\n", p);
        return;
    }
    if (p % 2 == 0) {
        printf("CPU: Is %d prime? false\n", p);
        return;
    }

    for (int i = 3; i * i <= p; i += 2) {
        if (p % i == 0) {
            printf("CPU: Is %d prime? false\n", p);
            return;
        }
    }
    printf("CPU: Is %d prime? true\n", p);
}

int main()
{
    int p;
    printf("Enter a number to test if it is prime: ");
    scanf("%d", &p);

    // Add vectors in parallel.
    hipError_t cudaStatus = isPrimeWithCuda(&p);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "isPrimeWithCuda failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // Check if the number is prime using CPU
    isPrimeCpu(p);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t isPrimeWithCuda(const int *p) {
    if (p == NULL) {
        fprintf(stderr, "Input number is NULL!");
        return hipErrorInvalidValue;
    }
    if (*p < 2) {
        fprintf(stderr, "Input number must be greater than 1!");
        return hipErrorInvalidValue;
    }
    if (*p == 2) {
        printf("Is %d prime? true\n", *p);
        return hipSuccess;
    }
    if (*p % 2 == 0) {
        printf("Is %d prime? false\n", *p);
        return hipSuccess;
    }

    int* dev_p = 0;
    bool* dev_isPrime = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for the input number and the output result.
    cudaStatus = hipMalloc((void**)&dev_p, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_isPrime, sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_p, p, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each a.
    millerRabinTestKernel<<<1, *p - 4>>>(dev_p, dev_isPrime);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "millerRabinTestKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching millerRabinTestKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    bool isPrime;
    cudaStatus = hipMemcpy(&isPrime, dev_isPrime, sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    printf("Is %d prime? %s\n", *p, isPrime ? "true" : "false");

Error:
    hipFree(dev_p);
    hipFree(dev_isPrime);

    return cudaStatus;
}
